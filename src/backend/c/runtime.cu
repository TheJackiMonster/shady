#include "hip/hip_runtime.h"
__shared__ uint3 __shady_make_thread_local(RealGlobalInvocationId);
__shared__ uint3 __shady_make_thread_local(RealLocalInvocationId);

#define GlobalInvocationId __shady_thread_local_access(RealGlobalInvocationId)
#define LocalInvocationId __shady_thread_local_access(RealLocalInvocationId)

__device__ void __shady_prepare_builtins() {
    LocalInvocationId.x = threadIdx.x;
    LocalInvocationId.y = threadIdx.y;
    LocalInvocationId.z = threadIdx.z;
    GlobalInvocationId.x = threadIdx.x + blockDim.x * blockIdx.x;
    GlobalInvocationId.y = threadIdx.y + blockDim.y * blockIdx.y;
    GlobalInvocationId.z = threadIdx.z + blockDim.z * blockIdx.z;
}

__device__ bool __shady_elect_first() {
    unsigned int writemask = __activemask();
    // Find the lowest-numbered active lane
    int elected_lane = __ffs(writemask) - 1;
    return threadIdx.x == __shfl_sync(writemask, threadIdx.x, elected_lane)
        && threadIdx.y == __shfl_sync(writemask, threadIdx.y, elected_lane)
        && threadIdx.z == __shfl_sync(writemask, threadIdx.z, elected_lane);
}

template<typename T>
__device__ T __shady_broadcast_first(T t) {
    unsigned int writemask = __activemask();
    // Find the lowest-numbered active lane
    int elected_lane = __ffs(writemask) - 1;
    return __shfl_sync(writemask, t, elected_lane);
}

__device__ static inline float sign(float f) {
    return copysignf(1.0f, f);
}